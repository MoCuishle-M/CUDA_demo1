#include <cstdio>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <random>

template <int blockSize>
__global__ void histogram(const int *hist_data, int *bin_data, int N) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // 初始化共享内存中的私有直方图
  extern __shared__ unsigned int histo_s[];
  for (unsigned int binIdx = threadIdx.x; binIdx < blockSize; binIdx += blockDim.x) {
	histo_s[binIdx] = 0u;
  }
  __syncthreads();

  // 计算局部直方图
  unsigned int prev_index = -1;
  unsigned int accumulator = 0;
  unsigned int curr_index;
  for (unsigned int i = tid; i < N; i += blockDim.x * gridDim.x) {
	unsigned char value = hist_data[i];
	curr_index = value;
	if (prev_index != curr_index) {
	  if (accumulator > 0)
		atomicAdd(&(histo_s[prev_index]), accumulator);
	  prev_index = curr_index;
	  accumulator = 1;
	} else {
	  accumulator++;
	}
  }

  if (accumulator > 0)
	atomicAdd(&(histo_s[prev_index]), accumulator);

  __syncthreads();

  // 合并局部直方图到全局内存
  for (unsigned int i = threadIdx.x; i < blockSize; i += blockDim.x) {
	atomicAdd(&(bin_data[i]), histo_s[i]);
  }
}

bool CheckResult(int *out, int* groudtruth, int N){
  for (int i = 0; i < N; i++){
	if (out[i] != groudtruth[i]) {
	  printf("in checkres, out[i]=%d, gt[i]=%d\n", out[i], groudtruth[i]);
	  return false;
	}
  }
  return true;
}

int main(){
  float milliseconds = 0;
  const int N = 25600000;
  int *h_hist = (int *)malloc(N * sizeof(int));
  int *bin = (int *)malloc(256 * sizeof(int));
  int *bin_data;
  int *hist_data;
  hipMalloc((void **)&bin_data, 256 * sizeof(int));
  hipMalloc((void **)&hist_data, N * sizeof(int));

  for(int i = 0; i < N; i++){
	h_hist[i] = i % 256;
  }
  unsigned seed = 37; // 你可以选择任何整数值作为种子
  std::mt19937 g(seed);
  std::shuffle(h_hist, h_hist+N, g);

  int *groudtruth = (int *)malloc(256 * sizeof(int));;
  for(int j = 0; j < 256; j++){
	groudtruth[j] = 100000;
  }

  hipMemcpy(hist_data, h_hist, N * sizeof(int), hipMemcpyHostToDevice);
  hipSetDevice(0);
  hipDeviceProp_t deviceProp{};
  hipGetDeviceProperties(&deviceProp, 0);
  const int blockSize = 256;
  int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
  dim3 Grid(GridSize);
  dim3 Block(blockSize);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  // bug1: L68的N不能传错，之前传的256，导致L19的cache[1]打印出来为0
  histogram<blockSize><<<Grid, Block>>>(hist_data, bin_data, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(bin, bin_data, 256 * sizeof(int), hipMemcpyDeviceToHost);
  // bug2: 同bug1，L67传进去的256表示两个buffer的数据量，这个必须得精确，之前传的N，尽管只打印第1个值，但依然导致L27打印出来的值为垃圾值
  bool is_right = CheckResult(bin, groudtruth, 256);
  if(is_right) {
	printf("the ans is right\n");
  } else {
	printf("the ans is wrong\n");
	for(int i = 0; i < 256; i++){
	  printf("%d ", bin[i]);
	}
	printf("\n");
  }
  printf("histogram + shared_mem + multi_value latency = %f ms\n", milliseconds);

  hipFree(bin_data);
  hipFree(hist_data);
  free(bin);
  free(h_hist);
  free(groudtruth);
}