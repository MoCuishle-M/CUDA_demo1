﻿#include <cstdio>
#include <hip/hip_runtime.h>


constexpr int kBlockSize = 256;
constexpr int kNumWaves = 1;
/*这里的意思就是我们还可以通过调整GridSize和BlockSize的方式获得更好的性能收益，也就是说一个线程负责更多的元素计算*/
int64_t GetNumBlocks(int64_t n) {
  int dev;
  {
	hipError_t err = hipGetDevice(&dev);
	if (err != hipSuccess) { return err; }
  }
  // SM的数量
  int sm_count;
  {
	hipError_t err = hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev);
	printf("sm_count: %d\n", sm_count);
	if (err != hipSuccess) { return err; }
  }
  // 每个SM的线程最大数量
  int tpm;
  {
	hipError_t err = hipDeviceGetAttribute(&tpm, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
	printf("Max Threads Per Multi Processor: %d\n", tpm);
	if (err != hipSuccess) { return err; }
  }
  //
  int64_t num_blocks = std::max<int64_t>(1, std::min<int64_t>((n + kBlockSize - 1) / kBlockSize,
															  sm_count * tpm / kBlockSize * kNumWaves));
  return num_blocks;
}

// 注意: v0-v5里面kernel得到的是各个block负责范围内的总和，要想得到最终的和，需要把各个block求得的总和再做reduce sum
// v6: multi-block reduce final result by two pass
template <int blockSize>
__device__ void BlockSharedMemReduce(float* smem) {
    //对v4 L45的for循环展开，以减去for循环中的加法指令，以及给编译器更多重排指令的空间
  if (blockSize >= 1024) {
    if (threadIdx.x < 512) {
      smem[threadIdx.x] += smem[threadIdx.x + 512];
    }
    __syncthreads();
  }
  if (blockSize >= 512) {
    if (threadIdx.x < 256) {
      smem[threadIdx.x] += smem[threadIdx.x + 256];
    }
    __syncthreads();
  }
  if (blockSize >= 256) {
    if (threadIdx.x < 128) {
      smem[threadIdx.x] += smem[threadIdx.x + 128];
    }
    __syncthreads();
  }
  if (blockSize >= 128) {
    if (threadIdx.x < 64) {
      smem[threadIdx.x] += smem[threadIdx.x + 64];
    }
    __syncthreads();
  }
  // the final warp
  if (threadIdx.x < 32) {
	volatile float* vshm = smem;
	float x = vshm[threadIdx.x];
	if (blockDim.x >= 64) {
	  x += vshm[threadIdx.x+32];__syncwarp();
	  vshm[threadIdx.x] = x;__syncwarp();
	}
	x += vshm[threadIdx.x+16];__syncwarp();
	vshm[threadIdx.x] = x;__syncwarp();
	x += vshm[threadIdx.x+8];__syncwarp();
	vshm[threadIdx.x] = x;__syncwarp();
	x += vshm[threadIdx.x+4];__syncwarp();
	vshm[threadIdx.x] = x;__syncwarp();
	x += vshm[threadIdx.x+2];__syncwarp();
	vshm[threadIdx.x] = x;__syncwarp();
	x += vshm[threadIdx.x+1];__syncwarp();
	vshm[threadIdx.x] = x;__syncwarp();
  }
}

template <int blockSize>
__global__ void reduce_v6(const float *d_in, float *d_out, int64_t nums){
    __shared__ float smem[blockSize];
    // 泛指当前线程在其block内的id
    unsigned int tid = threadIdx.x;
    // 泛指当前线程在所有block范围内的全局id
    unsigned int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int total_thread_num = blockDim.x * gridDim.x;
    // 基于v5的改进：不用显式指定一个线程处理2个元素，而是通过L94的for循环来自动确定每个线程处理的元素个数
    float sum = 0.0f;
    for (auto i = gtid; i < nums; i += total_thread_num) {
        sum += d_in[i];
    }
    smem[tid] = sum;
    __syncthreads();
    // compute: reduce in shared mem
    BlockSharedMemReduce<blockSize>(smem);

    // store: 哪里来回哪里去，把reduce结果写回显存
    // GridSize个block内部的reduce sum已得出，保存到d_out的每个索引位置
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(const float *out, float groundtruth, int n){
    if (*out != groundtruth) {
      return false;
    }
    return true;
}

int main(){
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, 0);
    int maxblocks = deviceProp.maxGridSize[0];
	printf("Device name: %s\n", deviceProp.name);
	printf("Maximum size of each dimension of a grid: %d\n", maxblocks);
    const int blockSize = 256;
    const int64_t N = 25600000;
    //int gridSize = std::min((N + blockSize - 1) / blockSize, maxblocks);

	// 通过v7中int64_t GetNumBlocks(int64_t n)函数得到的gridSize
  	const int64_t gridSize = GetNumBlocks(N);

    float milliseconds = 0;
    auto *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a,N * sizeof(float));

    auto *out = (float*)malloc((gridSize) * sizeof(float));
    float *d_out;
    float *part_out;//新增part_out存储每个block reduce的结果
    hipMalloc((void **)&d_out, 1 * sizeof(float));
    hipMalloc((void **)&part_out, (gridSize) * sizeof(float));
    auto ground_truth = static_cast<float>(N);

    for(int i = 0; i < N; i++){
        a[i] = 1;
    }

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(gridSize);
    dim3 Block(blockSize);
    

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    reduce_v6<blockSize><<<Grid, Block>>>(d_a, part_out, N);
    reduce_v6<blockSize><<<1, Block>>>(part_out, d_out, gridSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, 1 * sizeof(float), hipMemcpyDeviceToHost);
    bool is_right = CheckResult(out, ground_truth, 1);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0;i < 1;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }
    printf("reduce_v6 latency = %f ms\n", milliseconds);
  	printf("out[0]: %lf\n",out[0]);
  	printf("res[0]: %lf\n", ground_truth);
    hipFree(d_a);
    hipFree(d_out);
    hipFree(part_out);
    free(a);
    free(out);
}
