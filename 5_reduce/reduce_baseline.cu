#include <cstdio>
#include <hip/hip_runtime.h>

/*注: 每个cuda程序(.cu文件)的main函数功能大同小异，后面不会每个cu文件都注释main函数逻辑*/

__global__ void reduce_baseline(const int* input, int* output, size_t n) {

  int sum = 0;

  for (size_t i = 0; i < n; ++i) {
	sum += input[i];
  }

  *output = sum;
}

bool CheckResult(int *out, int groudtruth, int n){
  if (*out != groudtruth) {
	return false;
  }
  return true;
}

int main(){
  float milliseconds = 0;
  //const int N = 32 * 1024 * 1024;
  const int N = 25600000;
  hipSetDevice(0);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  //const int blockSize = 256;
  const int blockSize = 1;
  //int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);//used later
  int GridSize = 1;

  int *a = (int *)malloc(N * sizeof(int));
  int *d_a;
  hipMalloc((void **)&d_a, N * sizeof(int));

  int *out = (int*)malloc((GridSize) * sizeof(int));
  int *d_out;
  hipMalloc((void **)&d_out, (GridSize) * sizeof(int));

  for(int i = 0; i < N; i++){
	a[i] = 1;
  }

  int groudtruth = N * 1;

  hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);

  dim3 Grid(GridSize);
  dim3 Block(blockSize);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  reduce_baseline<<<1, 1>>>(d_a, d_out, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(out, d_out, GridSize * sizeof(int), hipMemcpyDeviceToHost);
  printf("allcated %d blocks, data counts are %d", GridSize, N);
  bool is_right = CheckResult(out, groudtruth, GridSize);
  if(is_right) {
	printf("the ans is right\n");
  } else {
	printf("the ans is wrong\n");
	for(int i = 0; i < GridSize;i++){
	  printf("res per block : %lf ",out[i]);
	}
	printf("\n");
	printf("groudtruth is: %f \n", groudtruth);
  }
  printf("reduce_baseline latency = %f ms\n", milliseconds);

  hipFree(d_a);
  hipFree(d_out);
  free(a);
  free(out);
}